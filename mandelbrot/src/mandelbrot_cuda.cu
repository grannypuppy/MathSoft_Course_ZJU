#include <hip/hip_runtime.h>

#include <stdio.h>
#include <vector>

// CUDA核函数，计算Mandelbrot集
__global__ void mandelbrotKernel(int* result, double xMin, double yMin, 
                               double xStep, double yStep, 
                               int width, int height, int maxIterations) {
    // 计算当前线程处理的坐标
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (x < width && y < height) {
        double real = xMin + x * xStep;
        double imag = yMin + y * yStep;
        
        // Mandelbrot迭代
        double zReal = 0;
        double zImag = 0;
        int iterations = 0;
        
        while (zReal * zReal + zImag * zImag <= 4.0 && iterations < maxIterations) {
            double tmp = zReal * zReal - zImag * zImag + real;
            zImag = 2.0 * zReal * zImag + imag;
            zReal = tmp;
            iterations++;
        }
        
        // 存储结果
        result[y * width + x] = iterations;
    }
}

// C++调用包装函数
extern "C" void computeMandelbrotCUDA(int* result, double xMin, double yMin, double xMax, double yMax,
                                    int width, int height, int maxIterations) {
    
    // 分配设备内存
    int* d_result;
    size_t size = width * height * sizeof(int);
    hipMalloc((void**)&d_result, size);
    
    // 计算步长
    double xStep = (xMax - xMin) / width;
    double yStep = (yMax - yMin) / height;
    
    // 配置CUDA网格和块
    dim3 blockSize(16, 16);
    dim3 gridSize((width + blockSize.x - 1) / blockSize.x, 
                 (height + blockSize.y - 1) / blockSize.y);
    
    // 启动CUDA核函数
    mandelbrotKernel<<<gridSize, blockSize>>>(d_result, xMin, yMin, xStep, yStep, 
                                            width, height, maxIterations);
    
    // 复制结果回主机
    hipMemcpy(result, d_result, size, hipMemcpyDeviceToHost);
    
    // 释放设备内存
    hipFree(d_result);
}